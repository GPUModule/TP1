#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include "hip/hip_runtime.h"
#include ""
#include "utils.h"

#define N 6
#define THREADS_PER_BLOCK 256
#define SQRT_THREADS_PER_BLOCK sqrt(THREADS_PER_BLOCK)

void checkCUDAError(const char*);
void random_floats(float *a, int n);
void print_array(float *a, int n, char *name);
int validate(float *a, float *ref, int n);

__global__ void simple_convolution2D_kernel(float* c, float* a, float* filter, int f, int n) {
  // A completer
}

__global__ void shared_convolution2D_kernel(float* c, float* a, float* filter, int f, int n) {
  // A completer
}

int main(void) {
	srand( time( NULL ) );

	float *a, *filter, *c;
	float *d_a, *d_filter, *d_c;
	int errors;
	
	//int f =
	//int n_c = 
	unsigned int filter_size = f * sizeof(float);
	unsigned int size = N * sizeof(float);
	unsigned int c_size = n_c * sizeof(float);

	event_pair timer;
	// Alloc space for device copies
	// A completer
	//hipMalloc
	//hipMalloc
	//hipMalloc
	checkCUDAError("CUDA malloc");

	// Alloc space for host copies
	// A completer
	//a = (float*)malloc(size);
	//filter = (float*)malloc(filter_size);
	//c = (float*)malloc(c_size);

	random_floats(a, N);
	random_floats(filter, f);
	print_array(a, N, "a");
	print_array(filter, f, "filter");

	// Copy inputs to device
	// A completer
	//hipMemcpy
	//hipMemcpy
	checkCUDAError("CUDA memcpy Host to Device");

	// Launch kernel on GPU
	dim3 blocksPerGrid((N + THREADS_PER_BLOCK -1)/THREADS_PER_BLOCK);
	dim3 threadsPerBlock(THREADS_PER_BLOCK);
	start_timer(&timer);
	simple_convolution2D_kernel<<<blocksPerGrid, threadsPerBlock>>>(d_c, d_a, d_filter, f, N);
	checkCUDAError("CUDA kernel");
	stop_timer(&timer,"Convolution 1D sur GPU");
	
	// Copy result back to host
	// A completer
	// hipMemcpy
	checkCUDAError("CUDA memcpy Device to Host");

	print_array(c, n_c, "c");
	
	//start_timer(&timer);
	//shared_convolution2D_kernel<<<blocksPerGrid, threadsPerBlock>>>(d_cs, d_a, d_filter, f, N);
	//checkCUDAError("CUDA kernel");
	//stop_timer(&timer,"Convolution 1D shared sur GPU");


	// validate
	//errors = validate(c, cs, n_c);
	//printf("CUDA GPU result has %d errors.\n", errors);

	// Cleanup
	free(a); free(filter); free(c);
	hipFree(d_a); hipFree(d_filter); hipFree(d_c);
	checkCUDAError("CUDA cleanup");

	return 0;
}

void checkCUDAError(const char *msg)
{
	hipError_t err = hipGetLastError();
	if (hipSuccess != err)
	{
		fprintf(stderr, "CUDA ERROR: %s: %s.\n", msg, hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
}

void random_floats(float *a, int n)
{
	for (unsigned int i = 0; i < n; i++){
			a[i] = (float)(rand() % 101);
	}
}

void print_array(float *a, int n, char*name){

	printf("%s : [ ",name);
	for (unsigned int i = 0; i < n; i++){
			printf("%.4f ",a[i]);
	}
	printf("]\n");
}

int validate(float *a, float *ref, int n){
	int errors = 0;
	for (unsigned int i = 0; i < n; i++){
		if (a[i] != ref[i]){
			errors++;
			fprintf(stderr, "ERROR at index %d: GPU result %f does not match CPU value of %f\n", i, a[i], ref[i]);
		}
	}

	return errors;
}